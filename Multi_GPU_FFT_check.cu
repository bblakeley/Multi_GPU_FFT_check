#include "hip/hip_runtime.h"
// Multiple GPU version of cuFFT_check that uses multiple GPU's
// This program creates a real-valued 3D function sin(x)*cos(y)*cos(z) and then 
// takes the forward and inverse Fourier Transform, with the necessary scaling included. 
// The output of this process should match the input function

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <complex.h>

// includes, project
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
//CUFFT Header file
#include <hipfft/hipfftXt.h>

#define NX 64
#define NY 64
#define NZ 64
#define NZ2 (NZ/2+1)
#define NN (NX*NY*NZ)
#define L (2*M_PI)
#define TX 8
#define TY 8
#define TZ 8

int divUp(int a, int b) { return (a + b - 1) / b; }

__device__
int idxClip(int idx, int idxMax){
    return idx > (idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);
}

__device__
int flatten(int col, int row, int stack, int width, int height, int depth){
    return idxClip(stack, depth) + idxClip(row, height)*depth + idxClip(col, width)*depth*height;
    // Note: using column-major indexing format
}

__global__ 
void initialize(int NX_per_GPU, int gpuNum, hipfftDoubleComplex *f1, hipfftDoubleComplex *f2)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;
    if ((i >= NX_per_GPU) || (j >= NY) || (k >= NZ)) return;
    const int idx = flatten(i, j, k, NX, NY, NZ);

    // Create physical vectors in temporary memory
    double x = i * (double)L / NX + (double)gpuNum*NX_per_GPU*L / NX;
    double y = j * (double)L / NY;
    double z = k * (double)L / NZ;

    // Initialize starting array
    f1[idx].x = sin(x)*cos(y)*cos(z);
    f1[idx].y = 0.0;

    f2[idx].x = 0.0;
    f2[idx].y = 0.0;

    return;
}

__global__
void scaleResult(int NX_per_GPU, hipfftDoubleComplex *f)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int k = blockIdx.z * blockDim.z + threadIdx.z;
    if ((i >= NX_per_GPU) || (j >= NY) || (k >= NZ)) return;
    const int idx = flatten(i, j, k, NX, NY, NZ);

    f[idx].x = f[idx].x / ( (double)NN );
    f[idx].y = f[idx].y / ( (double)NN );

    return;
}

int main (void)
{
    int i, j, k, idx, NX_per_GPU;
    // double complex test;

    // Set GPU's to use and list device properties
    int nGPUs = 2, deviceNum[nGPUs];
    for(i = 0; i<nGPUs; ++i)
    {
        deviceNum[i] = i;

        hipSetDevice(deviceNum[i]);

        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, deviceNum[i]);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    // Initialize input data
    // Split data according to number of GPUs
    NX_per_GPU = NX/nGPUs;              // This is not a good solution long-term; needs more work for arbitrary grid sizes/nGPUs

    // Declare variables
    hipfftDoubleComplex *u;
    hipfftDoubleComplex *u_fft;

    // Allocate memory for arrays
    hipMallocManaged(&u, sizeof(hipfftDoubleComplex)*NN );
    hipMallocManaged(&u_fft, sizeof(hipfftDoubleComplex)*NN );
    // Launch CUDA kernel to initialize velocity field
    const dim3 blockSize(TX, TY, TZ);
    const dim3 gridSize(divUp(NX_per_GPU, TX), divUp(NY, TY), divUp(NZ, TZ));
    for (i = 0; i<nGPUs; ++i){
        hipSetDevice(deviceNum[i]);
        int idx = i*NX_per_GPU*NY*NZ;                // sets the index value of the data to send to each gpu
        initialize<<<gridSize, blockSize>>>(NX_per_GPU, deviceNum[i], &u[idx], &u_fft[idx]);
    }

    // Synchronize both GPUs before moving forward
    for (i = 0; i<nGPUs; ++i){
        hipSetDevice(deviceNum[i]);
        hipDeviceSynchronize();
    }

    // Initialize CUFFT for multiple GPUs //
    // Initialize result variable used for error checking
    hipfftResult result;

    // Create empty plan that will be used for the FFT
    hipfftHandle plan;
    result = hipfftCreate(&plan);
    if (result != HIPFFT_SUCCESS) { printf ("*Create failed\n"); return 1; }

    // Tell cuFFT which GPUs to use
    result = hipfftXtSetGPUs (plan, nGPUs, deviceNum);
    if (result != HIPFFT_SUCCESS) { printf ("*XtSetGPUs failed\n"); return 1; }

    // Create the plan for the FFT
    size_t* worksize;                                   // Initializes the worksize variable
    worksize =(size_t*)malloc(sizeof(size_t) * nGPUs);  // Allocates memory for the worksize variable, which tells cufft how many GPUs it has to work with

    // Create the plan for cufft
    result = hipfftMakePlan3d(plan, NX, NY, NZ, HIPFFT_Z2Z, worksize);
    if (result != HIPFFT_SUCCESS) { printf ("*MakePlan* failed: code %d \n",(int)result); exit (EXIT_FAILURE) ; }

    // Initialize transform array - to be split among GPU's and transformed in place using cufftX
    hipLibXtDesc *u_prime;
    // Allocate data on multiple gpus using the cufft routines
    result = hipfftXtMalloc(plan, (hipLibXtDesc **)&u_prime, HIPFFT_XT_FORMAT_INPLACE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE) ; }

    // Copy the data from 'host' to device using cufftXt formatting
    result = hipfftXtMemcpy(plan, u_prime, u, HIPFFT_COPY_HOST_TO_DEVICE);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed, code: %d\n",result); exit (EXIT_FAILURE); }

    // Perform FFT on multiple GPUs
    printf("Forward 3d FFT on multiple GPUs\n");
    result = hipfftXtExecDescriptorZ2Z(plan, u_prime, u_prime, HIPFFT_FORWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExecZ2Z  failed\n"); exit (EXIT_FAILURE); }

////////// Apparently re-ordering the data prior to the IFFT is not necessary (gives incorrect results)////////////////////
    // hipLibXtDesc *u_reorder;
    // result = hipfftXtMalloc(plan, (hipLibXtDesc **)&u_reorder, HIPFFT_XT_FORMAT_INPLACE);
    // if (result != HIPFFT_SUCCESS) { printf ("*XtMalloc failed\n"); exit (EXIT_FAILURE) ; }
    // // Re-order data on multiple GPUs to natural order
    // printf("Reordering the data on the GPUs\n");
    // result = hipfftXtMemcpy (plan, u_reorder, u_prime, HIPFFT_COPY_DEVICE_TO_DEVICE);
    // if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE); }
/////////////////////////////////////////////////////////////////////////////////////////////

    // Perform inverse FFT on multiple GPUs
    printf("Inverse 3d FFT on multiple GPUs\n");
    result = hipfftXtExecDescriptorZ2Z(plan, u_prime,  u_prime, HIPFFT_BACKWARD);
    if (result != HIPFFT_SUCCESS) { printf ("*XtExecZ2Z  failed\n"); exit (EXIT_FAILURE); }

    // Copy the output data from multiple gpus to the 'host' result variable (automatically reorders the data from output to natural order)
    result = hipfftXtMemcpy (plan, u_fft, u_prime, HIPFFT_COPY_DEVICE_TO_HOST);
    if (result != HIPFFT_SUCCESS) { printf ("*XtMemcpy failed\n"); exit (EXIT_FAILURE); }

    // Scale output to match input (cuFFT does not automatically scale FFT output by 1/N)
    for (i = 0; i<nGPUs; ++i){
        hipSetDevice(deviceNum[i]);
        idx = i*NX_per_GPU*NY*NZ;                // sets the index value of the data to send to each gpu
        scaleResult<<<gridSize, blockSize>>>(NX_per_GPU, &u_fft[idx]);
    }

    // Synchronize GPUs
    for (i = 0; i<nGPUs; ++i){
        hipSetDevice(deviceNum[i]);
        hipDeviceSynchronize();
    }

    // Test results to make sure that u = u_fft
    double error = 0.0;
    for (i = 0; i<NX; ++i){
        for (j = 0; j<NY; ++j){
            for (k = 0; k<NZ; ++k){
                idx = k + j*NZ + NZ*NY*i;
                // error += (double)u[idx].x - sin(x)*cos(y)*cos(z);
                error += (double)u[idx].x - (double)u_fft[idx].x;
                // printf("At idx = %d, the value of the error is %f\n",idx,(double)u[idx].x - (double)u_fft[idx].x);
                // printf("At idx = %d, the value of the error is %f\n",idx,error);

            }
        }
    }
    printf("The sum of the errorx10^-7 is %f\n",error*10000000);

    // Deallocate variables

    // Free malloc'ed variables
    free(worksize);
    // Free cuda malloc'ed variables
    hipFree(u);
    hipFree(u_fft);
    // Free cufftX malloc'ed variables
    result = hipfftXtFree(u_prime);
    if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE); }
    // result = hipfftXtFree(u_reorder);
    // if (result != HIPFFT_SUCCESS) { printf ("*XtFree failed\n"); exit (EXIT_FAILURE); }
    // Destroy FFT plan
    result = hipfftDestroy(plan);
    if (result != HIPFFT_SUCCESS) { printf ("hipfftDestroy failed: code %d\n",(int)result); exit (EXIT_FAILURE); }

    return 0;

}